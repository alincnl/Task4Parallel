#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

using namespace std;
using namespace hipcub;

#define tol  1e-6

__global__ void update(double* A, double* Anew, int size)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (j < size - 1 && j > 0 && i > 0 && i < size - 1){
		double left = A[i * size + j - 1];
		double right = A[i * size + j + 1];
		double top = A[(i - 1) * size + j];
		double bottom = A[(i + 1) * size + j];
		Anew[i*size + j] = 0.25 * (left + right + top + bottom);
	}
}

__global__ void substract(double* A, double* Anew, double* res, int size){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i >= 0 && i < size && j >= 0 && j < size)
		res[i*size + j] = Anew[i*size + j] - A[i*size + j];
}

__constant__ double add;

__global__ void fill(double* A, double* Anew, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < size){
        A[i*(size) + 0] = 10 + add*i;
        A[i] = 10 + add*i;
        A[(size-1)*(size) + i] = 20 + add*i;
        A[i*(size)+size-1] = 20 + add*i;

        Anew[i*(size) + 0] = A[i*(size) + 0];
        Anew[i] = A[i];
        Anew[(size-1)*(size) + i] = A[(size-1)*(size) + i];
        Anew[i*(size)+size-1] = A[i*(size)+size-1];
    }
}

int main(int argc, char* argv[]){

    auto begin = std::chrono::steady_clock::now();
    hipSetDevice(1);
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    const int size =512, iter_max = 1000;

    double *d_A = NULL, *d_Anew = NULL, *d_Aprev;

    hipError_t cudaerr = hipSuccess;
    cudaerr = hipMalloc((void **)&d_A, sizeof(double)*size*size);
    if (cudaerr != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(cudaerr));
        exit(EXIT_FAILURE);
    }

    cudaerr = hipMalloc((void **)&d_Anew, sizeof(double)*size*size);
    if (cudaerr != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(cudaerr));
        exit(EXIT_FAILURE);
    }

    cudaerr = hipMalloc((void **)&d_Aprev, sizeof(double)*size*size);
    if (cudaerr != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(cudaerr));
        exit(EXIT_FAILURE);
    }

    int iter = 0;
    double error = 1;
    double addH = 10.0 / (size - 1);
    hipMemcpyToSymbol(HIP_SYMBOL(add), &addH, sizeof(double));

    dim3 threadPerBlock = dim3(32, 32);
    dim3 blocksPerGrid = dim3((size+threadPerBlock.x-1)/threadPerBlock.x,(size+threadPerBlock.y-1)/threadPerBlock.y);
    
    fill<<<blocksPerGrid, threadPerBlock>>>(d_A, d_Anew, size);

    double* d_error;
    hipMalloc(&d_error, sizeof(double));

    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_A, d_error, size*size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipStream_t stream;
    hipStreamCreate(&stream);

    bool graphCreated = false;
    hipGraph_t graph;
    hipGraphExec_t instance;

    while((error > tol) && (iter < iter_max/100)) {
        iter = iter + 2;
        if(!graphCreated)
	    {
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
            for(int i = 0; i<100;i++){
                update<<<blocksPerGrid, threadPerBlock,0,stream>>>(d_Anew,d_A, size);
                update<<<blocksPerGrid, threadPerBlock,0,stream>>>( d_A,  d_Anew,size);
            }
            hipStreamEndCapture(stream, &graph);
            hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
            graphCreated=true;
        }
       // swap = d_A;
       // d_Aprev=d_A;
        hipGraphLaunch(instance, stream);
	    hipStreamSynchronize(stream);

        substract<<<blocksPerGrid, threadPerBlock,0,stream>>>(d_A, d_Anew, d_Aprev, size);
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_Aprev, d_error, size*size,stream);
        hipMemcpyAsync(&error, d_error, sizeof(double), hipMemcpyDeviceToHost);
       // hipMemcpyAsync(d_A, d_Anew, size*size*sizeof(double), hipMemcpyDeviceToDevice);
        //swap = d_A;
        std::cout << iter << ":" << error << "\n";

    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time taken: %3.1f ms\n", elapsedTime);

    hipFree(d_A);
    hipFree(d_Anew);
    hipFree(d_error);

    auto end = std::chrono::steady_clock::now();
    auto elapsed_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end-begin);
    std::cout << "The time:" << elapsed_ms.count() << "ms\n";
    return 0;
}
